#include "hip/hip_runtime.h"
#include <cstdio> // std::printf, std::fflush, stdout
#include <vector> // std::vector<T>
#include <cassert> // assert

// #define DEBUG

#include "tfqmrgpu.hxx" // includes hip/hip_runtime.h and tfqmrgpu.h
#include "tfqmrgpu_linalg.hxx" // tfqmrgpu::transpose_blocks_kernel
#include "tfqmrgpu_core.hxx"   // tfqmrgpu::solve<action_t>
#include "tfqmrgpu_blocksparse.hxx" // blocksparse_action_t
#include "tfqmrgpu_util.hxx" // IgnoreCase


    template <typename real_t, int LM, int LN=LM, typename double_t=real_t>
    tfqmrgpuStatus_t mysolve_real_LM_LN (
          hipStream_t streamId // stream
        , bsrsv_plan_t* p // plan
        , double const tolerance
        , int const MaxIterations
        , bool const memcount
    ) {
        blocksparse_action_t<real_t,LM,LN,double_t> action(p);
        return tfqmrgpu::solve(action, memcount ? nullptr : p->pBuffer, tolerance, MaxIterations, streamId);
    } // mysolve_real_LM_LN


    template <int LM, int LN=LM>
    tfqmrgpuStatus_t mysolve_LM_LN (
          hipStream_t streamId // stream
        , bsrsv_plan_t* p // plan
        , double const tolerance
        , int const MaxIterations
        , bool const memcount
    ) {
        switch (p->doublePrecision | IgnoreCase) {
          case 'z': return mysolve_real_LM_LN<double,LM,LN>(streamId, p, tolerance, MaxIterations, memcount);
          case 'm': return mysolve_real_LM_LN<float,LM,LN,double>(streamId, p, tolerance, MaxIterations, memcount); // mixed precision: load float, multipy-accumulate double, store float
          default : return mysolve_real_LM_LN<float,LM,LN>(streamId, p, tolerance, MaxIterations, memcount);
        }
    } // mysolve_LM_LN


    tfqmrgpuStatus_t mysolve (
          hipStream_t streamId
        , bsrsv_plan_t* p // plan
        , double const tolerance
        , int const MaxIterations
        , bool const memcount=false
    ) {
        switch (p->LM*1000 + p->LN) {
#define     allow_block_size(LM,LN) \
            case   LM*1000 + LN: return mysolve_LM_LN<LM,LN>(streamId, p, tolerance, MaxIterations, memcount)

            // list all the allowed block sizes here as allow_block_size(ldA, ldB);
#include    "allowed_block_sizes.h"
//          allow_block_size( 4, 4);
//          allow_block_size( 8, 8);
//          allow_block_size( 8,32); // blocks in X and B are rectangular
//          allow_block_size(16,16);
//          allow_block_size(32,32);
//          allow_block_size(64,64);

#undef      allow_block_size
            default: return TFQMRGPU_BLOCKSIZE_MISSING + TFQMRGPU_CODE_CHAR*p->LM + TFQMRGPU_CODE_LINE*p->LN; // also say which blocksize was requested
        } // switch LM
    } // mysolve





    // library peripherals ////////////////////////////////////////
    template <typename T>
    T tfqmrgpu_mem_align(T a) { return (((a - 1) >> TFQMRGPU_MEMORY_ALIGNMENT) + 1) << TFQMRGPU_MEMORY_ALIGNMENT; }

    tfqmrgpuStatus_t tfqmrgpuPrintError(tfqmrgpuStatus_t const status) {
        tfqmrgpuStatus_t stat{status};
        char const key = stat / TFQMRGPU_CODE_CHAR;
                  stat -= key * TFQMRGPU_CODE_CHAR;
        uint32_t const line = stat / TFQMRGPU_CODE_LINE;
                  stat -= line * TFQMRGPU_CODE_LINE;
        switch (stat) {
            case TFQMRGPU_STATUS_SUCCESS:          debug_printf("tfQMRgpu: Success!\n");           break;
            case TFQMRGPU_STATUS_ALLOCATION_FAILED: std::printf("tfQMRgpu: Allocation failed!\n"); break;
            case TFQMRGPU_POINTER_INVALID:          std::printf("tfQMRgpu: Pointer invalid!\n");   break;
            case TFQMRGPU_STATUS_MAX_ITERATIONS:    std::printf("tfQMRgpu: Max number of iterations exceeded!\n");       break;
            case TFQMRGPU_STATUS_BREAKDOWN:         std::printf("tfQMRgpu: All components have broken down!\n");         break;
            case TFQMRGPU_NO_IMPLEMENTATION:        std::printf("tfQMRgpu: Missing implementation at line %d!\n", line); break;
            case TFQMRGPU_UNDOCUMENTED_ERROR:       std::printf("tfQMRgpu: Undocumented error at line %d!\n",     line); break;
            case TFQMRGPU_BLOCKSIZE_MISSING:        std::printf("tfQMRgpu: Missing blocksize %d x %d!\n",              key, line); break;
            case TFQMRGPU_TANSPOSITION_UNKNOWN:     std::printf("tfQMRgpu: Unknown transposition '%c' at line %d!\n",  key, line); break;
            case TFQMRGPU_VARIABLENAME_UNKNOWN:     std::printf("tfQMRgpu: Unknown variable name '%c' at line %d!\n",  key, line); break;
            case TFQMRGPU_DATALAYOUT_UNKNOWN:       std::printf("tfQMRgpu: Unknown data layout '%c' at line %d!\n", 20+key, line); break;
            case TFQMRGPU_PRECISION_MISSMATCH:      std::printf("tfQMRgpu: Missmatch in precision '%c' at line %d!\n", key, line); break;
            default:                                std::printf("tfQMRgpu: Unknown status= %d at line %d!\n", status, line); break;
        } // switch status
        std::fflush(stdout);
        return TFQMRGPU_STATUS_SUCCESS;
    } // printError

    tfqmrgpuStatus_t tfqmrgpuCreateHandle(tfqmrgpuHandle_t *handle) { // out: opaque handle for the tfqmrgpu library.
        if (nullptr != *handle) return TFQMRGPU_UNDOCUMENTED_ERROR + TFQMRGPU_CODE_LINE*__LINE__;
        *handle = (tfqmrgpuHandle_t) new tfq_handle_t(); // create new and cast pointer
        return (nullptr != *handle)? TFQMRGPU_STATUS_SUCCESS : TFQMRGPU_STATUS_ALLOCATION_FAILED;
    } // createHandle

    tfqmrgpuStatus_t tfqmrgpuDestroyHandle(tfqmrgpuHandle_t handle) { // inout: opaque handle for the tfqmrgpu library.
        if (nullptr == handle) return TFQMRGPU_UNDOCUMENTED_ERROR + TFQMRGPU_CODE_LINE*__LINE__;
        delete (tfq_handle_t*) handle; // cast pointer and delete
        return TFQMRGPU_STATUS_SUCCESS;
    } // destroyHandle


    tfqmrgpuStatus_t tfqmrgpuSetStream(tfqmrgpuHandle_t handle, // inout: opaque handle for the tfqmrgpu library.
        hipStream_t const streamId) { // in: GPU stream to be used by tfqmrgpu
        ((tfq_handle_t*) handle)->streamId = streamId;
        return TFQMRGPU_STATUS_SUCCESS;
    } // setStream

    tfqmrgpuStatus_t tfqmrgpuGetStream(tfqmrgpuHandle_t handle, // in: opaque handle for the tfqmrgpu library.
        hipStream_t      *streamId) { // out: GPU stream used by tfqmrgpu
        *streamId = ((tfq_handle_t*) handle)->streamId;
        return TFQMRGPU_STATUS_SUCCESS;
    } // getStream

    tfqmrgpuStatus_t tfqmrgpu_bsrsv_createPlan(
          tfqmrgpuHandle_t handle // none: opaque handle for the tfqmrgpu library.
        , tfqmrgpuBsrsvPlan_t *plan // out: newly created plan
        , int     const mb          // in: number of block rows in A, X and B == number of block columns in A
        , int32_t const *bsrRowPtrA // in: integer array of mb+1 elements that contains the start of every block row of A and the end of the last block row of A plus one.
        , int     const nnzbA       // in: number of nonzero blocks of matrix A
        , int32_t const *bsrColIndA // in: integer array of nnzbA ( = bsrRowPtrA[mb] - bsrRowPtrA[0] ) column indices of the nonzero blocks of matrix A.
        , int32_t const *bsrRowPtrX // in: integer array of mb+1 elements that contains the start of every block row of X and the end of the last block row of X plus one.
        , int     const nnzbX       // in: number of nonzero blocks of matrix X
        , int32_t const *bsrColIndX // in: integer array of nnzbX ( = bsrRowPtrX[mb] - bsrRowPtrX[0] ) column indices of the nonzero blocks of matrix X.
        , int32_t const *bsrRowPtrB // in: integer array of mb+1 elements that contains the start of every block row of B and the end of the last block row of B plus one.
        , int     const nnzbB       // in: number of nonzero blocks of matrix B, nnzbB must be less or equal to nnzbX.
        , int32_t const *bsrColIndB // in: integer array of nnzbB ( = bsrRowPtrB[mb] - bsrRowPtrB[0] ) column indices of the nonzero blocks of matrix B.
        , int     const indexOffset // in: indexOffset=0(C-style) or indexOffset=1(Fortran) for RowPtr and ColInd arrays    
    ) {
        debug_printf("tfqmrgpu_bsrsv_createPlan(handle=%p, *plan=%p, mb=%d, \n"
               "         bsrRowPtrA=%p, nnzbA=%d, bsrColIndA=%p, \n"
               "         bsrRowPtrX=%p, nnzbX=%d, bsrColIndX=%p, \n"
               "         bsrRowPtrB=%p, nnzbB=%d, bsrColIndB=%p, indexOffset=%d)\n",
               handle, *plan, mb,             bsrRowPtrA, nnzbA, bsrColIndA, 
               bsrRowPtrX, nnzbX, bsrColIndX, bsrRowPtrB, nnzbB, bsrColIndB, indexOffset);
        std::fflush(stdout);

        if (nullptr != *plan)   return TFQMRGPU_UNDOCUMENTED_ERROR + TFQMRGPU_CODE_LINE*__LINE__; // requirement that *plan == nullptr on entry.

        // compute Y = A*X, minimize |Y - B| to solve A*X == B

        // static plausibility checks
        if (mb < 1)             return TFQMRGPU_UNDOCUMENTED_ERROR + TFQMRGPU_CODE_LINE*__LINE__; // at least one row/column needs to be there.
        if (nnzbX < 1)          return TFQMRGPU_UNDOCUMENTED_ERROR + TFQMRGPU_CODE_LINE*__LINE__; // at least one block of X needs to be found.
        if (nnzbB > nnzbX)      return TFQMRGPU_UNDOCUMENTED_ERROR + TFQMRGPU_CODE_LINE*__LINE__; // the non-zero pattern of B must be a true subset of that of X.
        if (nnzbA > mb*mb)      return TFQMRGPU_UNDOCUMENTED_ERROR + TFQMRGPU_CODE_LINE*__LINE__; // the operator A is assumed logically square, mb*mb is the upper bound.
        if (nnzbA != bsrRowPtrA[mb] - bsrRowPtrA[0])  return TFQMRGPU_UNDOCUMENTED_ERROR + TFQMRGPU_CODE_LINE*__LINE__; // the operator A is not sane
        if (nnzbX != bsrRowPtrX[mb] - bsrRowPtrX[0])  return TFQMRGPU_UNDOCUMENTED_ERROR + TFQMRGPU_CODE_LINE*__LINE__; // the operator X is not sane
        if (nnzbB != bsrRowPtrB[mb] - bsrRowPtrB[0])  return TFQMRGPU_UNDOCUMENTED_ERROR + TFQMRGPU_CODE_LINE*__LINE__; // the operator B is not sane

        bsrsv_plan_t* const p = new bsrsv_plan_t(); // allocate the plan in host memory
        p->cpu_mem = sizeof(bsrsv_plan_t); // init host memory usage with the memory capacity required by the struct itself
        p->gpu_mem = 0; // initialize for GPU memory count

        p->nRows = mb;
        p->nnzbA = nnzbA;

        int const C0F1 = indexOffset; // abbreviate start index C/C++:0, Fortran:1

        { // in this scope we compute the multiplication index pair list for Y = A * X

            // the BSR sparsity pattern of Y is equal to the BSR sparsity pattern of X
            auto const bsrRowPtrY = bsrRowPtrX; // copy pointer
            auto const bsrColIndY = bsrColIndX; // copy pointer

            auto const nnzbY = nnzbX; // copy number of non-zero elements
            size_t const estimate_n_pairs = (nnzbY * nnzbA) / mb; // approximate number of block operations
            debug_printf("tfqmrgpu_bsrsv_createPlan tries to reserve %ld pairs\n", estimate_n_pairs);
            p->pairs.clear();
            p->pairs.reserve(2 * estimate_n_pairs); // factor 2 as we always save pairs of indices

            p->starts.clear();
            p->starts.reserve(nnzbY + 1); // exact size

            for (auto irow = 0; irow < mb; ++irow) {
                for (auto inzy = bsrRowPtrY[irow] - C0F1; inzy < bsrRowPtrY[irow + 1] - C0F1; ++inzy) {
                    auto const jcol = bsrColIndY[inzy]; // warning, jcol starts from 1 in Fortran
                    // now compute Y[irow][jcol] = sum_k A[irow][kcol] * X[krow][jcol] with k==kcol==krow

                    p->starts.push_back(p->pairs.size()/2);

                    for (auto inza = bsrRowPtrA[irow] - C0F1; inza < bsrRowPtrA[irow + 1] - C0F1; ++inza) {
                        auto const kcol = bsrColIndA[inza] - C0F1;
                        auto const krow = kcol;
                        assert(krow >= 0); assert(krow < mb);
                        auto const inzx = find_in_array(bsrRowPtrX[krow] - C0F1, // begin
                                                        bsrRowPtrX[krow + 1] - C0F1, // end
                                                        jcol, // try to find this value
                                                        bsrColIndX); // in this array
                        if (inzx >= 0) {
                            p->pairs.push_back(inza);
                            p->pairs.push_back(inzx);
                        } // exists
                    } // inza
                } // inzy
            } // irow

            // this last entry is very important for the sparse matrix format
            p->starts.push_back(p->pairs.size()/2);

            assert(nnzbY + 1 == p->starts.size()); // sanity check

            debug_printf("# found %ld pairs in A*X multiplication\n", p->pairs.size()/2); // log output

            p->pairs.shrink_to_fit(); // free unused host memory
#ifdef DEBUG
            std::printf("# p->pairs.data()  = %p\n", (char*)(p->pairs.data()));
            std::printf("# p->starts.data() = %p\n", (char*)(p->starts.data()));
#endif // DEBUG
            p->cpu_mem += p->starts.size() * sizeof(uint32_t); // register host memory usage in Byte
            p->cpu_mem += p->pairs.size()  * sizeof(uint32_t); // register host memory usage in Byte
        } // scope


        { // in this scope we check if B is a true subset of X
          // and compute the sparse subset list for operations of type X -= B or X += B
            p->subset.clear();
            p->subset.reserve(nnzbB); // exact size
            for (auto irow = 0; irow < mb; ++irow) {
                for (auto inzb = bsrRowPtrB[irow] - C0F1; inzb < bsrRowPtrB[irow + 1] - C0F1; ++inzb) {
                    auto const inzx = find_in_array(bsrRowPtrX[irow] - C0F1, // begin
                                                    bsrRowPtrX[irow + 1] - C0F1, // end
                                                    bsrColIndB[inzb], // try to find this value
                                                    bsrColIndX); // in this array
                    if (inzx < 0) return TFQMRGPU_UNDOCUMENTED_ERROR + TFQMRGPU_CODE_LINE*__LINE__; // B is not a true subset of X
                    p->subset.push_back(inzx); // store the block index into the value array of X at which B is also non-zero.
                } // inzb
            } // irow
            assert(nnzbB == p->subset.size()); // sanity check

            p->cpu_mem += p->subset.size() * sizeof(uint32_t); // register host memory usage in Byte
        } // scope


        { // in this scope we try to find the number of block columns in X and B
          // and we create a compressed copy of the bsrColIndX list called colindx

            int32_t min_colInd = 2e9, max_colInd = -min_colInd; // init close to the largest int32_t
            for (auto inzx = 0; inzx < nnzbX; ++inzx) {
                auto const jcol = bsrColIndX[inzx]; // we do not need to subtract the Fortran 1 here.
                min_colInd = std::min(min_colInd, jcol); // find the minimum index
                max_colInd = std::max(max_colInd, jcol); // find the maxmimum index
            } // inzx
            auto const nc = 1 + max_colInd - min_colInd; // preliminary number of columns computed via the range of indices
            if (nc < 1) return TFQMRGPU_UNDOCUMENTED_ERROR + TFQMRGPU_CODE_LINE*__LINE__; // at least one column must be in X and B

            // check which indices in the range [min_colInd, max_colInd] are touched
            std::vector<uint32_t> nRowsPerColX(nc, 0);
            for (auto inzx = 0; inzx < nnzbX; ++inzx) {
                auto const jcol = bsrColIndX[inzx];
                auto const jc = jcol - min_colInd;
                assert(jc >= 0);
                ++nRowsPerColX[jc];
            } // inzx

            std::vector<int32_t> translate_jc2jb(nc);
            unsigned nempty{0}, nb{0}; // number of block columns
            for (auto jc = 0; jc < nc; ++jc) {
                if (0 == nRowsPerColX[jc]) {
                    translate_jc2jb[jc] = -1; // empty column
                    ++nempty;
                } else {
                    translate_jc2jb[jc] = nb; // valid column
                    ++nb;
                }
            } // jc
            // now nb is the number of block columns after filtering out the empty columns

            // warn if there are empty columns as these should be erased before. Is erasing really necessary?
            if (nempty > 0) {
                debug_printf("# found %d columns without non-zero entries!\n", nempty); // warning output
            } // nempty

            p->colindx.clear();
            p->colindx.resize(nnzbX); // exact size

            p->original_bsrColIndX.clear();
            p->original_bsrColIndX.resize(nb); // exact size

            for (auto inzx = 0; inzx < nnzbX; ++inzx) {
                auto const jcol = bsrColIndX[inzx];
                auto const jc = jcol - min_colInd; // jc in [0, nc)
                assert(jc >= 0); assert(jc < nc);
                auto const jb = translate_jc2jb[jc]; // jb in [0, nb)
                assert(jb >= 0); assert(jb < nb);
                p->colindx[inzx] = jb; // or p->colindx.push_back(jb); // but then we need reserve instead of resize above
                p->original_bsrColIndX[jb] = jcol; // retrieval information for debugging
            } // inzx

            p->cpu_mem += p->colindx.size() * sizeof(uint16_t); // register host memory usage in Byte
            p->cpu_mem += p->original_bsrColIndX.size() * sizeof(int32_t); // register host memory usage in Byte
            p->nCols = nb; // store number of block columns
        } // scope

        p->pBuffer = nullptr; // init pointer copy to device memory (which will be allocated by the user)

        p->flops_performed_all = 0; // init
        p->flops_performed    = -1; // init impossible
        p->iterations_needed  = -1; // init impossible

        debug_printf("# found %ld non-zero entries in X\n", p->colindx.size());
        assert(p->colindx.size() == nnzbX);

        *plan = (tfqmrgpuBsrsvPlan_t) p; // cast into opaque pointer type

        debug_printf("done tfqmrgpu_bsrsv_createPlan(handle=%p, *plan=%p, [internal p=%p] ...)\n", handle, *plan, p);
        return TFQMRGPU_STATUS_SUCCESS;
    } // analysis

    tfqmrgpuStatus_t tfqmrgpu_bsrsv_destroyPlan(
          tfqmrgpuHandle_t handle  // none: opaque handle for the tfqmrgpu library.
        , tfqmrgpuBsrsvPlan_t plan // plan is destroyed
    ) {
        auto const p = (bsrsv_plan_t const*) plan;
        if (nullptr == p) return TFQMRGPU_POINTER_INVALID;
        delete p;
        return TFQMRGPU_STATUS_SUCCESS;
    } // destroyPlan


    tfqmrgpuStatus_t tfqmrgpu_bsrsv_bufferSize(
          tfqmrgpuHandle_t handle // in: opaque handle for the tfqmrgpu library.
        , tfqmrgpuBsrsvPlan_t plan // inout: plan becomes enriched by LM and doublePrecision
        , int const ldA         // in: Leading dimension for blocks in matrix A.
        , int const blockDim    // in: Block dimension of matrix A, blocks in A are square blocks. blockDim <= ldA
        , int const ldB         // in: Leading dimension for blocks in matrix B or X.
        , int const RhsBlockDim // in: Fast block dimension of matrix B or X
        , char const doublePrecision // in: Solver precision 'C':complex<float>, 'Z':complex<double>, 'M':load float and compute double.
        , size_t *pBufferSizeInBytes // out: number of bytes of the buffer used in setMatrix, getMatrix and solve.
    ) {
        // query the necessary GPU memory buffer size
        int const LM = ldA;
        int const LN = ldB;
        if (LM != blockDim)     return TFQMRGPU_UNDOCUMENTED_ERROR + TFQMRGPU_CODE_LINE*__LINE__; // so far, this library is not that flexible
        if (LM > LN)            return TFQMRGPU_UNDOCUMENTED_ERROR + TFQMRGPU_CODE_LINE*__LINE__; // so far, this library is not that flexible
        if (LN != RhsBlockDim)  return TFQMRGPU_UNDOCUMENTED_ERROR + TFQMRGPU_CODE_LINE*__LINE__; // so far, this library is not that flexible

        auto const p = (bsrsv_plan_t*)plan;

        switch (doublePrecision | IgnoreCase) {
            case 'f':
            case 'c': p->doublePrecision = 'c'; break;  // single precision complex
            case 'm': p->doublePrecision = 'm'; break;  // mixed  precision complex, ToDo: test
            case 'd':
            case 'z': p->doublePrecision = 'z'; break;  // double precision complex
            default : p->doublePrecision = 'z'; // default double precision complex
        } // doublePrecision
        if (doublePrecision != p->doublePrecision) {
            debug_printf("# convert doublePrecision= \'%c\' to \'%c\'\n", doublePrecision, p->doublePrecision);
        }

        p->LM = LM; // store the block size and precision information in the plan
        p->LN = LN; // store the number of columns in each block of X or B

        hipStream_t streamId;
        {   auto const stat = tfqmrgpuGetStream(handle, &streamId);
            if (TFQMRGPU_STATUS_SUCCESS != stat) return stat;
        }

        if (nullptr == pBufferSizeInBytes) return TFQMRGPU_UNDOCUMENTED_ERROR + TFQMRGPU_CODE_LINE*__LINE__;

        bool const memcount = true;
        auto const status = mysolve(streamId, p, 0.0, 0, memcount); // call the solver in memcount-mode

        *pBufferSizeInBytes = p->gpu_mem; // requested minimum number of Bytes in device memory
        debug_printf("# plan for doublePrecision= \'%c\' and LM= %d, LN= %d needs %.3f MByte device memory\n",
                              p->doublePrecision,         p->LM,  p->LN,    p->gpu_mem*1e-6);
        return status;
    } // bufferSize


    tfqmrgpuStatus_t tfqmrgpu_bsrsv_setBuffer(
          tfqmrgpuHandle_t handle // in: opaque handle for the tfqmrgpu library.
        , tfqmrgpuBsrsvPlan_t plan // inout: set the plan-internal buffer variable
        , void* const pBuffer // in: pointer to GPU memory
    ) {
        if (nullptr == pBuffer) return TFQMRGPU_UNDOCUMENTED_ERROR + TFQMRGPU_CODE_LINE*__LINE__;

        hipStream_t streamId{0};
        {   auto const stat = tfqmrgpuGetStream(handle, &streamId);
            if (TFQMRGPU_STATUS_SUCCESS != stat) return stat;
        }

        auto const p = (bsrsv_plan_t*) plan;
        p->pBuffer = (char*)pBuffer; // buffer setting

        { // random number generation scope
            auto const n_floats_in_v3 = p->vec3win.length/sizeof(float);
            auto const v3 = (float*)(p->pBuffer + p->vec3win.offset);
            debug_printf("# v3 has address %p\n", (void*)v3);
            auto const stat = tfqmrgpu::create_random_numbers(v3, n_floats_in_v3, streamId);
            {
                float first{0}, flast{0};
                get_data_from_gpu<float>(&first, v3, 1, 0, "first of v3");
                get_data_from_gpu<float>(&flast, &v3[n_floats_in_v3 - 1], 1, 0, "last of v3");
                debug_printf("# v3 has values %g ... %g\n", first, flast);
            }
            if (TFQMRGPU_STATUS_SUCCESS != stat) return stat;
        } // scope

        { // in this scope we transfer the integer vectors 
          // that are filled during the analysis step.
            tfqmrgpu::transfer_index_lists(streamId, p);
        } // scope

        return TFQMRGPU_STATUS_SUCCESS;
    } // setBuffer

    // registers the GPU memory buffer pointer in the handle and calls the random number generator.
    tfqmrgpuStatus_t tfqmrgpu_bsrsv_getBuffer(
          tfqmrgpuHandle_t handle // none: opaque handle for the tfqmrgpu library.
        , tfqmrgpuBsrsvPlan_t plan // in: plan for bsrsv, read the plan-internal buffer variable
        , void* *pBuffer // out: pointer to GPU memory
    ) {
        auto const p = (bsrsv_plan_t const*) plan;
        *pBuffer = (void*)p->pBuffer;
        if (nullptr == *pBuffer) return TFQMRGPU_UNDOCUMENTED_ERROR + TFQMRGPU_CODE_LINE*__LINE__;
        return TFQMRGPU_STATUS_SUCCESS;
    } // getBuffer

namespace tfqmrgpu {

    // asynchronous setting/getting of matrix operands
    tfqmrgpuStatus_t set_or_getMatrix(
          tfqmrgpuHandle_t handle // in: opaque handle for the tfqmrgpu library.
        , tfqmrgpuBsrsvPlan_t plan // inout: plan for bsrsv
        , char const var // in: selector which variable, only {'A', 'X', 'B'} allowed.
        , char const* const values_in  // pointer to read-only values, pointer is casted to float* or double*
        , char const doublePrecision='z' // in: 'C','c':complex<float>, 'Z','z':complex<double>, 's' and 'd' are not supported.
        , char const transposition='n' // in: transposition of the input matrix blocks.
        , tfqmrgpuDataLayout_t const layout=TFQMRGPU_LAYOUT_RIRIRIRI
        , char       *const values_out=nullptr   // pointer to values, pointer is casted to float* or double*
    ) {
        bool const is_get = (nullptr != values_out);
        debug_printf("# tfqmrgpu::%cetMatrix for operator \'%c\', values=%p\n", is_get?'g':'s', var, is_get?values_out:values_in);

        {
            switch (layout) {
                case TFQMRGPU_LAYOUT_RRRRIIII: break; // native for this GPU solver
                case TFQMRGPU_LAYOUT_RIRIRIRI: break; // native for e.g. Fortran complex arrays
                case TFQMRGPU_LAYOUT_RRIIRRII: break; // Beware: not tested
                default: return TFQMRGPU_DATALAYOUT_UNKNOWN + TFQMRGPU_CODE_CHAR*layout + TFQMRGPU_CODE_LINE*__LINE__;
            } // switch layout
        }

        double scal_imag{1};
        char trans = transposition | IgnoreCase; // non-const copy
        {
            switch (trans) {
                case 'h':
                case 'c': scal_imag = -1; trans = 't'; break; // transpose + conjugate // LAPACK uses 'c' for the Hermitian adjoint, but allow also 'H' or 'h'
                case '*': scal_imag = -1; trans = 'n'; break; //        only conjugate
                case 't': break; // transpose
                case 'n': break; // non-transpose
                default: return TFQMRGPU_TANSPOSITION_UNKNOWN + TFQMRGPU_CODE_CHAR*trans + TFQMRGPU_CODE_LINE*__LINE__;
            } // switch trans
            assert('n' == trans || 't' == trans);
        }

        auto const p = (bsrsv_plan_t const*) plan;
        uint32_t nnzb{0}, nRows{p->LM}, nCols{p->LN};
        char* ptr = is_get ? nullptr : p->pBuffer;
        size_t size{0}; // size in Byte
        {
            switch (var | IgnoreCase) {
                case 'a':
                    ptr += p->matAwin.offset;
                    size = p->matAwin.length;
                    nnzb = p->nnzbA;
                    nCols = p->LM;
                    // internally, operator A is stored column major for coalesced memory access on the GPU
                    if ('n' == trans) { trans = 't'; } else // this flip could be written as trans = int('n') + int('t') - trans;
                    if ('t' == trans) { trans = 'n'; } else
                    { return TFQMRGPU_TANSPOSITION_UNKNOWN + TFQMRGPU_CODE_CHAR*trans + TFQMRGPU_CODE_LINE*__LINE__; }
                    debug_printf("# tfqmrgpu_bsrsv_setMatrix: flip transposition "
                      "'%c' to internal '%c' for operator '%c'\n", transposition, trans, var);
                break;
                case 'b':
                    ptr += p->matBwin.offset;
                    size = p->matBwin.length;
                    nnzb = p->subset.size();
                break;
                case 'x':
                    ptr += p->matXwin.offset;
                    size = p->matXwin.length;
                    nnzb = p->colindx.size();
                break;
                // the passed variable name does not carry a meaning
                default: return TFQMRGPU_VARIABLENAME_UNKNOWN + TFQMRGPU_CODE_CHAR*var + TFQMRGPU_CODE_LINE*__LINE__; 
            } // switch var
        }
        if (nnzb < 1) return TFQMRGPU_STATUS_SUCCESS; // nothing to do
        assert(nullptr != ptr);

        auto const dp = ('z' == p->doublePrecision);
        if (('z' == (doublePrecision | IgnoreCase)) != dp) {
            std::printf("# mismatch: \'%c\' and plan says \'%c\'\n", doublePrecision, p->doublePrecision);
            return TFQMRGPU_PRECISION_MISSMATCH + TFQMRGPU_CODE_CHAR*doublePrecision + TFQMRGPU_CODE_LINE*__LINE__;
        }

        auto const byte_per_block = 2*nRows*nCols*(dp ? sizeof(double) : sizeof(float));
        assert(nnzb*byte_per_block == size);

        hipStream_t streamId{0};
        {   auto const stat = tfqmrgpuGetStream(handle, &streamId);
            if (TFQMRGPU_STATUS_SUCCESS != stat) return stat;
        }

        bool trans_in{false}, trans_out{false};
        tfqmrgpuDataLayout_t l_in, l_out;
        if (is_get) {
            trans_in = ('t' == trans);
            l_in = TFQMRGPU_LAYOUT_RRRRIIII;
            l_out = layout;
        } else {
            assert(nullptr != values_in);
            l_in = layout;
            l_out = TFQMRGPU_LAYOUT_RRRRIIII;
            trans_out = ('t' == trans);
            debug_printf("# start asynchronous memory transfer from the host to the GPU for operator '%c'\n", var);
            copy_data_to_gpu<char>(ptr, values_in, size, streamId);
            debug_printf("#  done asynchronous memory transfer from the host to the GPU for operator '%c'\n", var);
        } // get or set

        // for each block change data layout and (if necessary) transpose in-place on the GPU
        if (dp) {
            tfqmrgpu::transpose_blocks_kernel<double>
#ifndef HAS_NO_CUDA
                <<< nnzb, {nCols,nRows,1}, byte_per_block, streamId >>>
#endif // HAS_CUDA
                ((double*)ptr, nnzb, 1, scal_imag, l_in, l_out, trans_in, trans_out, nRows, nCols, var);
        } else {
            tfqmrgpu::transpose_blocks_kernel<float>
#ifndef HAS_NO_CUDA
                <<< nnzb, {nCols,nRows,1}, byte_per_block, streamId >>>
#endif // HAS_CUDA
                ((float *)ptr, nnzb, 1, scal_imag, l_in, l_out, trans_in, trans_out, nRows, nCols, var);
        } // dp

        if (is_get) {
            debug_printf("# start asynchronous memory transfer from the GPU to the host for operator '%c'\n", var);
            get_data_from_gpu<char>(values_out, ptr, size, streamId);
            debug_printf("#  done asynchronous memory transfer from the GPU to the host for operator '%c'\n", var);
        } // get

        return TFQMRGPU_STATUS_SUCCESS;
    } // set_or_getMatrix

} // namespace tfqmrgpu


    // asynchronous setting of matrix operands, C-interface
    tfqmrgpuStatus_t tfqmrgpu_bsrsv_setMatrix(
          tfqmrgpuHandle_t handle // in: opaque handle for the tfqmrgpu library.
        , tfqmrgpuBsrsvPlan_t plan // inout: plan for bsrsv
        , char const var // in: selector which variable, only {'A', 'X', 'B'} allowed.
        , void const *const values // in: pointer to read-only values, pointer is casted to float* or double*
        , char const doublePrecision // in: 'c':complex<float>, 'z':complex<double>, 's' and 'd' are not supported.
        , int const ld // in: leading dimension of blocks in array values, not in use.
        , int const d2 // in:  second dimension of blocks in array values, not in use.
        , char const trans // in: transposition of the input matrix blocks.
        , tfqmrgpuDataLayout_t const layout
    ) {
        return tfqmrgpu::set_or_getMatrix(handle, plan, var, (char const*)values, doublePrecision, trans, layout);
    } // setMatrix

    // download of matrix operands, C-interface
    tfqmrgpuStatus_t tfqmrgpu_bsrsv_getMatrix(
          tfqmrgpuHandle_t handle // in: opaque handle for the tfqmrgpu library.
        , tfqmrgpuBsrsvPlan_t plan // in: plan for bsrsv
        , char const var // in: selector which variable, only 'X' or 'x' supported.
        , void       *const values // out: pointer to writeable values, pointer is casted to float* or double*
        , char const doublePrecision // in: 'c':complex<float>, 'z':complex<double>, 's' and 'd' are not supported.
        , int const ld // in: leading dimension of blocks in array values, not in use.
        , int const d2 // in:  second dimension of blocks in array values, not in use.
        , char const trans // in: transposition of the output matrix blocks.
        , tfqmrgpuDataLayout_t const layout
    ) {
        if ('x' != (var | IgnoreCase)) {
            // Only the download of operator 'X' is allowed.
            // Internally, operator A is stored column major, so downloading in e.g. with trans 'n'
            // would first modify the value of the operator A in-place on the GPU, so
            // solving again, e.g. with a modified right hand side B might lead to unexpected results
            // therefore, we do not allow downloading of operator A
            // similarly, B, therefore, we only allow to download operator X
            return TFQMRGPU_UNDOCUMENTED_ERROR + TFQMRGPU_CODE_CHAR*var + TFQMRGPU_CODE_LINE*__LINE__;
        } // only operator A
        return tfqmrgpu::set_or_getMatrix(handle, plan, var, 0x0, doublePrecision, trans, layout, (char *)values);
    } // getMatrix


    tfqmrgpuStatus_t tfqmrgpu_bsrsv_solve(
          tfqmrgpuHandle_t handle // in: opaque handle for the tfqmrgpu library.
        , tfqmrgpuBsrsvPlan_t plan // inout: plan for bsrsv
        , double const threshold // in: convergence threshold
        , int const maxIterations // in: maximum number of solver iterations
    ) {
        hipStream_t streamId;
        {   auto const stat = tfqmrgpuGetStream(handle, &streamId);
            if (TFQMRGPU_STATUS_SUCCESS != stat) return stat;
        }
        auto const p = (bsrsv_plan_t*) plan;

        return mysolve(streamId, p, threshold, maxIterations);
    } // solve (wrapper)

    tfqmrgpuStatus_t tfqmrgpu_bsrsv_getInfo(
          tfqmrgpuHandle_t handle // in: no function
        , tfqmrgpuBsrsvPlan_t plan // in: contains state
        , double *residuum_reached // out: residuum after iterations
        , int32_t *iterations_needed // out: number of iterations needed to converge
        , double *flops_performed // out: number of floating pointer operations performed for the last run
        , double *flops_performed_all // out: number of floating pointer operations performed since createPlan
    ) {
        auto const p = (bsrsv_plan_t const*) plan; // convert opaque plan object
        int any{0};
        if (nullptr != residuum_reached   ) { ++any; *residuum_reached    = p->residuum_reached; }
        if (nullptr != iterations_needed  ) { ++any; *iterations_needed   = p->iterations_needed; }
        if (nullptr != flops_performed    ) { ++any; *flops_performed     = p->flops_performed;    }
        if (nullptr != flops_performed_all) { ++any; *flops_performed_all = p->flops_performed_all; }

        return any ? TFQMRGPU_STATUS_SUCCESS : TFQMRGPU_STATUS_NO_INFO_PASSED;
    } // getInfo

    // utilities for the Fortran interface
    tfqmrgpuStatus_t tfqmrgpuCreateWorkspace(
          void* *pBuffer
        , size_t const pBufferSizeInBytes
        , char const MemoryType
    ) {
        hipError_t err;
        if ('m' == (MemoryType | IgnoreCase)) { // 'm' or 'M' stand for "managed"
            err = hipMallocManaged(pBuffer, pBufferSizeInBytes);
        } else {
            err = hipMalloc(pBuffer, pBufferSizeInBytes);
        }
        return (hipSuccess == err) ? TFQMRGPU_STATUS_SUCCESS : TFQMRGPU_STATUS_ALLOCATION_FAILED;
    } // createWorkspace

    tfqmrgpuStatus_t tfqmrgpuDestroyWorkspace(void* pBuffer) {
        return hipFree(pBuffer);
    } // destroyWorkspace


    tfqmrgpuStatus_t tfqmrgpu_bsrsv_z(
          int const mb // number of block rows and block columns in A
        , int const ldA // number of rows in a block
        , int const ldB // number of columns in a block of X or B
        , int32_t const *const rowPtrA
        , int const nnzbA
        , int32_t const *const colIndA
        , double  const *const Amat // assumed data layout double A[nnzbA][ldA][ldA][2]
        , char const transA
        , int32_t const *const rowPtrX
        , int const nnzbX
        , int32_t const *const colIndX
        , double        *const Xmat // assumed data layout double X[nnzbX][ldA][ldB][2]
        , char const transX
        , int32_t const *const rowPtrB
        , int const nnzbB
        , int32_t const *const colIndB
        , double  const *const Bmat // assumed data layout double B[nnzbB][ldA][ldB][2]
        , char const transB
        , int32_t *const iterations // on entry: max. number of iterations, on exit: needed number of iterations
        , float *const residual // on entry: required residuum for convergence, on exit: residdum reached
        , int const echo // verbosity to stdout
    ) {
        if (echo > 0) std::printf("# %s: mb= %d, ldA= %d, ldB= %d, iterations= %d, residual= %.1e\n", __func__, mb, ldA, ldB, *iterations, *residual);
        tfqmrgpuHandle_t handle{0};
        auto stat = tfqmrgpuCreateHandle(&handle);
        if (stat) { if (echo > 0) std::printf("# %s: tfqmrgpuCreateHandle returned %d\n", __func__, stat); return stat; }

        stat = tfqmrgpuSetStream(handle, 0); // set default stream
        if (stat) { if (echo > 0) std::printf("# %s: tfqmrgpuSetStream returned %d\n", __func__, stat); return stat; }

        tfqmrgpuBsrsvPlan_t plan{0};
        stat = tfqmrgpu_bsrsv_createPlan(handle, &plan, mb
                                  , rowPtrA, nnzbA, colIndA
                                  , rowPtrX, nnzbX, colIndX
                                  , rowPtrB, nnzbB, colIndB, 0);
        if (stat) { if (echo > 0) std::printf("# %s: tfqmrgpu_bsrsv_createPlan returned %d\n", __func__, stat); return stat; }

        size_t gpu_memory_size{0};
        stat = tfqmrgpu_bsrsv_bufferSize(handle, plan, ldA, ldA, ldB, ldB, 'z', &gpu_memory_size);
        if (stat) { if (echo > 0) std::printf("# %s: tfqmrgpu_bsrsv_bufferSize returned %d\n", __func__, stat); return stat; }

        void* gpu_memory_buffer{nullptr};
        stat = tfqmrgpuCreateWorkspace(&gpu_memory_buffer, gpu_memory_size, 'd'); // device memory
        if (stat) {
            if (echo > 0) std::printf("# %s: tfqmrgpuCreateWorkspace returned %d\n", __func__, stat);
            if (echo > 3) std::printf("# %s: probably running on hardware without GPUs\n", __func__);
            return stat;
        } // stat

        stat = tfqmrgpu_bsrsv_setBuffer(handle, plan, gpu_memory_buffer);
        if (stat) { if (echo > 0) std::printf("# %s: tfqmrgpu_bsrsv_setBuffer returned %d\n", __func__, stat); return stat; }

        stat = tfqmrgpu_bsrsv_setMatrix(handle, plan, 'A', Amat, 'z', ldA, ldA, transA, TFQMRGPU_LAYOUT_RIRIRIRI);
        if (stat) { if (echo > 0) std::printf("# %s: tfqmrgpu_bsrsv_setMatrix(\'A\') returned %d\n", __func__, stat); return stat; }

        stat = tfqmrgpu_bsrsv_setMatrix(handle, plan, 'B', Bmat, 'z', ldB, ldA, transB, TFQMRGPU_LAYOUT_RIRIRIRI);
        if (stat) { if (echo > 0) std::printf("# %s: tfqmrgpu_bsrsv_setMatrix(\'B\') returned %d\n", __func__, stat); return stat; }

        double const threshold = (nullptr != residual) ? *residual : 1e-9;
        int const maxiter = (nullptr != iterations) ? *iterations : 200;
        stat = tfqmrgpu_bsrsv_solve(handle, plan, threshold, maxiter);
        if (stat) { if (echo > 0) std::printf("# %s: tfqmrgpu_bsrsv_solve returned %d\n", __func__, stat); return stat; }

        double residuum{0}, flops{0}, flops_all{0};
        int32_t needed{0};
        stat = tfqmrgpu_bsrsv_getInfo(handle, plan, &residuum, &needed, &flops, &flops_all);
        if (stat) { if (echo > 0) std::printf("# %s: tfqmrgpu_bsrsv_getInfo returned %d\n", __func__, stat); return stat; }
        if (echo > 1) std::printf("# tfQMRgpu needed %d iterations to converge to %.1e using %g GFlop\n", needed, residuum, flops*1e-9);
        if (nullptr != residual) *residual = residuum;
        if (nullptr != iterations) *iterations = needed;

        stat = tfqmrgpu_bsrsv_getMatrix(handle, plan, 'X', Xmat, 'z', ldB, ldA, transX, TFQMRGPU_LAYOUT_RIRIRIRI);
        if (stat) { if (echo > 0) std::printf("# %s: tfqmrgpu_bsrsv_getMatrix returned %d\n", __func__, stat); return stat; }

        stat = tfqmrgpuDestroyWorkspace(gpu_memory_buffer);
        if (stat) { if (echo > 0) std::printf("# %s: tfqmrgpuDestroyWorkspace returned %d\n", __func__, stat); return stat; }

        stat = tfqmrgpu_bsrsv_destroyPlan(handle, plan);
        if (stat) { if (echo > 0) std::printf("# %s: tfqmrgpu_bsrsv_destroyPlan returned %d\n", __func__, stat); return stat; }

        stat = tfqmrgpuDestroyHandle(handle);
        if (stat) { if (echo > 0) std::printf("# %s: tfqmrgpuDestroyHandle returned %d\n", __func__, stat); return stat; }

        return TFQMRGPU_STATUS_SUCCESS;
    } // tfqmrgpu_bsrsv_z
