#include "hip/hip_runtime.h"
#include <cstdio> // std::printf
#include <cstdlib> // std::atoi
#include <iostream> // std::cout, ::endl
#include <fstream> // std::ifstream
#include <algorithm> // std::max
#include <cmath> // std::sqrt
#include <vector> // std::vector<T>
#include <cassert> // assert

// #define DEBUG

#include "tfqmrgpu.hxx" // includes hip/hip_runtime.h (or tfqmrgpu_cudaStubs.hxx) and tfqmrgpu.h
#include "bsr.hxx" // bsr_t [block-sparse row matrices]
#include "tfqmrgpu_example_reader.hxx" // ::read_in()
#include "tfqmrgpu_example_xml_reader.hxx" // ::read_in()

#include "tfqmrgpu_util.hxx" // FlopChar, CCheck, copy_data_to_gpu, get_data_from_gpu, IgnoreCase
#ifndef HAS_NO_CUDA
    #include "tfqmrgpu_blockmult.hxx" // gemmNxNf
#endif // HAS_CUDA

#ifdef DEBUG
    #define debug_printf(...) std::printf(__VA_ARGS__)
#else  // DEBUG
    #define debug_printf(...)
#endif // DEBUG


namespace GPUbench {

    // Example routine using the tfQMRgpu library's C-interface

    int benchmark_tfQMRgpu_library(
          bsr_t const ABX[3]
        , double const tolerance=1.0e-6
        , int const maxIterations=999
        , int const nRepetitions=1
        , char const precision='z' // {'c', 'z'}
    ) {

        PUSH_RANGE(__func__); // NVTX range markers for nvvp
        std::printf("\n# %s on GPU !!!!\n", __func__);

        auto const A = &(ABX[0]), B = &(ABX[1]), X = &(ABX[2]); // abbreviations

        int const lm = A->fastBlockDim; // number of rows per block
        int const ln = X->fastBlockDim; // number of cols per block
        assert ( ln == B->fastBlockDim );
        assert ( lm == X->slowBlockDim );
        assert ( lm == B->slowBlockDim );
        assert ( lm == A->slowBlockDim && "Blocks of A must be square" );

#define callAndCheck(FUN) \
        { \
            debug_printf("\n# Start "#FUN"\n"); \
            auto const stat = FUN; \
            debug_printf("# Done  "#FUN"\n"); \
            tfqmrgpuPrintError(stat); \
            if (TFQMRGPU_STATUS_SUCCESS != stat) return stat; \
        }

        // step 1: create a handle for the function call
        tfqmrgpuHandle_t handle{0};
        callAndCheck(  tfqmrgpuCreateHandle(&handle)  )

        // step 2: create a CUDA stream to work on
        hipStream_t streamId{0};
        if (1) {
            auto const cudaErr = hipStreamCreate(&streamId);
            if (hipSuccess != cudaErr) {
                std::printf("[ERROR] CUDA call failed to create a stream in %s:%d\n", __FILE__, __LINE__);
                return TFQMRGPU_UNDOCUMENTED_ERROR + TFQMRGPU_CODE_LINE*__LINE__;
            }
        } else { std::printf("# Warning! GPU works on default stream!\n"); }

        // step 3: register the CUDA stream in the handle
        callAndCheck(  tfqmrgpuSetStream(handle, streamId)  )

        if (1) { // sanity check
            auto streamId_copy{streamId};
            callAndCheck(  tfqmrgpuGetStream(handle, &streamId_copy)  )
            assert(streamId == streamId_copy);
        } // sanity check

        // step 4: analyse the blocks-sparse-row matrix patterns and create a bsrsv-plan
        tfqmrgpuBsrsvPlan_t plan{0};
        std::printf("\n# nnzb for A=%d, X=%d, B=%d\n", A->nnzb, X->nnzb, B->nnzb);
        callAndCheck(  tfqmrgpu_bsrsv_createPlan(handle, &plan, 
           A->nRows, // the number of block rows in A, X and B, also the number of block columns in A
           A->RowPtr.data(), A->nnzb, A->ColInd.data(), // block-sparse-row structure for A
           X->RowPtr.data(), X->nnzb, X->ColInd.data(), // block-sparse-row structure for X
           B->RowPtr.data(), B->nnzb, B->ColInd.data(), // block-sparse-row structure for B
           0, 9)  ) // indexOffset=0(C-style) or indexOffset=1(Fortran), echo=9

        // step 5: compute the GPU memory requirement based on block sizes
        size_t pBufferSize{0}; // in Bytes
        std::printf("# compute the GPU memory requirements for precision=\'%c\' LM=%d LN=%d\n", precision, lm, ln);
        callAndCheck(  tfqmrgpu_bsrsv_bufferSize(handle, plan,
            lm, // Leading dimension for blocks in matrix A.
            lm, // Block dimension of matrix A, blocks in A are square blocks. lm <= ldA
            ln, // Leading dimension for blocks in matrix B or X.
            ln, // Fast block dimension of matrix B or X, RhsBlockDim <= ldB.
            precision, // Solver precision 'c':complex<float>, 'z':complex<double>, 'm':mixed
            &pBufferSize)  )

        // step 6: allocate the GPU memory
        void* pBuffer{nullptr};
        {
            auto const cudaErr = hipMalloc(&pBuffer, pBufferSize);
            if (hipSuccess != cudaErr) {
                std::printf("[ERROR] CUDA call failed to allocate %.6f GByte in %s:%d\n", pBufferSize*1e-9, __FILE__, __LINE__);
                return TFQMRGPU_STATUS_ALLOCATION_FAILED + TFQMRGPU_CODE_LINE*__LINE__;
            } else {
                debug_printf("# allocated %.6f GByte GPU memory at %p in %s:%d\n", pBufferSize*1e-9, pBuffer, __FILE__, __LINE__);
                std::printf("# use %.6f GByte GPU memory\n", pBufferSize*1e-9);
            }
        }

        // step 7: register the GPU memory buffer in the bsrsv-plan
        callAndCheck(  tfqmrgpu_bsrsv_setBuffer(handle, plan, pBuffer)  )

        if (1) { // sanity check
            auto pBuffer_copy{pBuffer};
            callAndCheck(  tfqmrgpu_bsrsv_getBuffer(handle, plan, &pBuffer_copy)  )
            assert(pBuffer == pBuffer_copy);
        } // sanity check

        // create array for the result
        auto const sizeX = X->mat.size();
        std::vector<double> result(sizeX, 0.0);

        // step 8a: upload the values for the matrix A
        // values come from Fortran, so they are in RIRIRIRI layout
        std::vector<float> Amat_float, Bmat_float, Xmat_float;
        auto Amat = (char*)A->mat.data();
        auto Bmat = (char*)B->mat.data();
        auto Xmat = (char*)result.data();
        if ('z' != (precision | IgnoreCase)) {
            Amat_float.resize(A->mat.size());
            for (size_t bij = 0; bij < A->mat.size(); ++bij) Amat_float[bij] = A->mat[bij]; // convert to float
            Amat = (char*)Amat_float.data();

            Bmat_float.resize(B->mat.size());
            for (size_t bij = 0; bij < B->mat.size(); ++bij) Bmat_float[bij] = B->mat[bij]; // convert to float
            Bmat = (char*)Bmat_float.data();

            Xmat_float.resize(sizeX, 0.f);
            Xmat = (char*)Xmat_float.data();
        } // single precision

        callAndCheck(  tfqmrgpu_bsrsv_setMatrix(handle, plan, 'A', Amat, precision, lm, lm, 't', TFQMRGPU_LAYOUT_RIRIRIRI)  )

        // step 8b: upload the values for the right-hand side vectors B
        // values come from Fortran, so we need to transpose the blocks of B
        callAndCheck(  tfqmrgpu_bsrsv_setMatrix(handle, plan, 'B', Bmat, precision, ln, lm, 't', TFQMRGPU_LAYOUT_RIRIRIRI)  )

        // [optional] step 8x: upload the values for the initial vectors X

        // step 9: envoke the transpose-free Quasi Minimal Residual solver
        double solver_time = - getTime(); // start timer
        callAndCheck(  tfqmrgpu_bsrsv_solve(handle, plan, tolerance, maxIterations)  )
        solver_time += getTime(); // stop timer

        // step a: spare line
        // step b: spare line
        // step c: spare line


        // step d: retrieve the result vectors X 
        // convert the blocks into ColMajor and RIRIRIRI to match the Fortran data layout
        callAndCheck(  tfqmrgpu_bsrsv_getMatrix(handle, plan, 'X', Xmat, precision, ln, lm, 't', TFQMRGPU_LAYOUT_RIRIRIRI)  )
        if (Xmat_float.data() == (float*)Xmat) {
            for (size_t bij = 0; bij < X->mat.size(); ++bij) result[bij] = Xmat_float[bij]; // convert to double
        } // single precision

        { // scope:
            PUSH_RANGE("compare@CPU");
            double alldev{0}, allval{0}, maxdev{0}, maxrel{0};
            double const *const reference = X->mat.data();
            for(auto cij = 0ull; cij < sizeX; ++cij) {
                double const dev = std::abs(result[cij] - reference[cij]);
                maxdev = std::max(maxdev, dev);
                if (0.0 != reference[cij]) maxrel = std::max(maxrel, dev/reference[cij]);
                alldev += dev;
                allval += 1.0;
            } // cij
            std::printf("# GPU maxdev %g avgdev %g maxrel %g\n", maxdev, alldev/allval, maxrel);
            POP_RANGE(); // end of NVTX range

            if (maxdev < 1e-5) {
                // seems correct, report performance
                int iterations_needed{0};
                double flops_performed{0}, residuum_reached{1};
                callAndCheck(  tfqmrgpu_bsrsv_getInfo(handle, plan, &residuum_reached,
                                            &iterations_needed, &flops_performed, 0x0)
                            )
                std::printf("# GPU converged to %.1e in %d iterations\n", residuum_reached, iterations_needed);
                char const fF = ('z' == (precision | IgnoreCase))? 'F' : 'f'; // 'F':double, 'f':float
                auto const TFlop = 1e-12*flops_performed;
                auto const performance = TFlop/std::max(solver_time, 1e-6);
                std::printf("# GPU performed %.3f T%clop in %.3f seconds = %.3f T%clop/s\n",
                                            TFlop, fF, solver_time, performance, fF);
            } // maxdev
        } // scope

        // step e: destroy the plan
        callAndCheck(  tfqmrgpu_bsrsv_destroyPlan(handle, plan)  )
        plan = 0;

        // step f: destroy the handle
        callAndCheck(  tfqmrgpuDestroyHandle(handle)  )
        handle = 0;

        // last step, free GPU memory
        CCheck(hipFree(pBuffer));

#undef  callAndCheck
        POP_RANGE(); // end of NVTX range
        return TFQMRGPU_STATUS_SUCCESS;
    } // benchmark_tfQMRgpu_library
















    // Multiplication benchmark: measure the performance of Y = A*X

    template <typename T>
    T* get_gpu_memory(size_t const size=1) {
#ifdef DEBUGGPU
        std::printf("#  hipMalloc: %lu x %.3f kByte = \t%.3E MByte", size, 1e-3*sizeof(T), size*1e-6*sizeof(T));
#endif // DEBUGGPU
        void* d = nullptr;
        CCheck(hipMalloc(&d, size*sizeof(T)));
#ifdef DEBUGGPU
        std::printf(" @ %p through %p \n", d, (char*)d + size*sizeof(T) - 1);
#endif // DEBUGGPU
        return (T*)d;
    } // get_gpu_memory

    template <typename T>
    void free_gpu_memory(T*& d) {
        CCheck(hipFree(d));
        d = nullptr;
    } // free_gpu_memory

    template <typename T>
    T* create_on_gpu(T const *const h, size_t const size=1, hipStream_t const stream=0) {
        T* d = get_gpu_memory<T>(size);
        copy_data_to_gpu<T>(d, h, size, stream); // start copying to the GPU, async!
        return d;
    } // create_on_gpu

    template <typename T>
    T* create_on_cpu(T const (*devPtr d), size_t const size=1, hipStream_t const stream=0) {
        T* h = (T*) malloc(size*sizeof(T)); // c-style allocation
        get_data_from_gpu<T>(h, d, size, stream); // start copying from the GPU, async!
        return h;
    } // create_on_cpu

#ifndef HAS_NO_CUDA
    template <typename real_t, int LM, int LN>
    void __global__ // GPU kernel, must be launched with <<< {nmat, 1, 1}, {LN, any, 1} >>>
    fill_cos_sin(real_t (*devPtr c)[2][LM][LN]) {
        // fill GPU arrays with non-trivial but deterministic values
        int const m = blockIdx.x;
        int const j = threadIdx.x;
        for(int i = threadIdx.y; i < LM; i += blockDim.y) { // grid stride loop
            auto const arg = double((m*LM + i)*LN + j);
            c[m][0][i][j] = std::cos(arg);
            c[m][1][i][j] = std::sin(arg);
        } // i
    } // fill_cos_sin
#endif // HAS_CUDA

    template <typename real_t, int LM, int LN=LM, typename double_t=real_t, int TUNE=2>
    double bench_multi( // returns the average time needed per kernel call
          unsigned const nnzbY
        , uint32_t const starts_h[]
        , size_t const nPairs
        , uint32_t const pairs_h[]
        , unsigned const nnzbA
        , unsigned const nnzbX
        , int const nRepetitions=1 // Number of iterations of the same procedure
        , int const nSamples=1 // Number of samples taken for timings
    ) {
        std::printf("\n# %s<%d,%d> on GPU !!!!\n", __func__, LM, LN);

        std::printf("# Execute %d repetitions, sample %d times.\n", nRepetitions, nSamples);

        size_t const mem = (nnzbY + nnzbX)*sizeof(real_t[2][LM][LN]) + nnzbA*sizeof(real_t[2][LM][LM]);
        std::printf("# Try to allocate %.3f GByte for %d complex matrices of dim=%d x %d and "
          "%d + %d complex matrices of dim=%d x %d\n", mem*1e-9, nnzbA, LM, LM, nnzbY, nnzbX, LM, LN);
        auto matA = get_gpu_memory<real_t[2][LM][LM]>(nnzbA);
        auto matX = get_gpu_memory<real_t[2][LM][LN]>(nnzbX);
        auto matY = get_gpu_memory<real_t[2][LM][LN]>(nnzbY);

        auto starts_d = create_on_gpu<uint32_t>(starts_h, nnzbY + 1);
        auto pairs_d  = create_on_gpu<uint32_t>(pairs_h, nPairs*2);

#ifndef HAS_NO_CUDA
        fill_cos_sin<real_t,LM,LM> <<< nnzbA, {LM, 1024/LM, 1} >>> (matA);
        fill_cos_sin<real_t,LM,LN> <<< nnzbX, {LN, 1024/LN, 1} >>> (matX);

        // TUNE == 2 performance up to 3.8 TFlop/s for LM=32 on V100
        // TUNE == 4 performance up to 4.3 TFlop/s for LM=32 on V100, does not work for LM=6
        dim3 const threads = { LN, TUNE, 1 };
        std::printf("# CUDA Launch <<< %d, { %d, %d, %d } >>>, TUNE = %d\n",
                              nnzbY, threads.x, threads.y, threads.z, TUNE);
#endif // HAS_CUDA
        assert(nnzbX == nnzbY); // operator A must be logically square

        double nFlop{0};
        double time_sum{0}, time_rms{0}; // timing stats
        PUSH_RANGE("GPU benchmarks gemmNxNf");
        for(int sample = 0; sample < nSamples; ++sample) {
            double time{-getTime()}; // start
            for(int repetition = 0; repetition < nRepetitions; ++repetition) {
                // test the small matrix-matrix multiplications
#ifndef HAS_NO_CUDA
                gemmNxNf<real_t,LM,LN,LM/TUNE,double_t> <<< nnzbY, threads >>> (matY, matA, matX, pairs_d, starts_d);
                nFlop += nPairs*(8.*LM)*(LM*LN);
#endif // HAS_CUDA
            } // repetition
            CCheck(hipDeviceSynchronize());
            time += getTime(); // stop
            time_sum += time; time_rms += time*time; // add to timing stats
        } // sample
        POP_RANGE(); // end of NVTX range

        double const time_avg = time_sum/nSamples; // average
        time_rms = std::sqrt(std::max(0., time_rms/nSamples - time_avg*time_avg)); // rms
        std::printf("# GPU needed %.3f seconds, %.6f +/- %.6f sec per sample, %.1f%% dev\n",
                        time_sum, time_avg, time_rms, time_rms*100./time_avg);

        bool correct{true};
#ifdef  SKIP_CORRECTNESS_CHECK
        std::printf("# Warning! Correctness checks are deactivated with -D SKIP_CORRECTNESS_CHECK!\n");
#else // SKIP_CORRECTNESS_CHECK
        PUSH_RANGE("CPU checks correctness");
        double time_chk =- getTime();
        // check if matY has the correct values
        double maxdev={-1}, alldev{0}, allval{0};
        int nthreads{1};
#pragma omp parallel
        { nthreads = omp_get_num_threads(); }
        std::printf("# CPU %d threads check for correct results\n", nthreads);
        { // correctness check scope
            auto const matA_h = create_on_cpu<real_t[2][LM][LM]>(matA, nnzbA);
            auto const matX_h = create_on_cpu<real_t[2][LM][LN]>(matX, nnzbX);
            auto const matY_h = create_on_cpu<real_t[2][LM][LN]>(matY, nnzbY);
#pragma omp parallel for reduction(+:alldev,allval) reduction(max:maxdev)
            for(auto iY = 0u; iY < nnzbY; ++iY) {
                auto const matY_r = new double_t[2][LM][LN]; // thread-private reference result
                for(int i = 0; i < LM; ++i) {
                    for(int j = 0; j < LN; ++j) {
                        matY_r[0][i][j] = 0; // clear real part
                        matY_r[1][i][j] = 0; // clear imaginary part
                    } // j
                } // i
                for(auto ipair = starts_h[iY]; ipair < starts_h[iY + 1]; ++ipair) {
                    auto const iA = pairs_h[ipair*2 + 0], iX = pairs_h[ipair*2 + 1];
                    for(int i = 0; i < LM; ++i) {
                        for(int j = 0; j < LN; ++j) {
                            double_t cr{0}, ci{0};
                            for(int k = 0; k < LM; ++k) {
                                // matA is stored in transposed fashion in tfQMRgpu
                                double_t const srei = matA_h[iA][0][k][i],
                                               simi = matA_h[iA][1][k][i];
                                double_t const vrej = matX_h[iX][0][k][j],
                                               vimj = matX_h[iX][1][k][j];
                                cr += srei * vrej - simi * vimj; // Real part
                                ci += srei * vimj + simi * vrej; // Imag part
                            } // k
                            matY_r[0][i][j] += cr;
                            matY_r[1][i][j] += ci;
                        } // j
                    } // i
                } // ipair
                for(int c = 0; c < 2; ++c) {
                    for(int i = 0; i < LM; ++i) {
                        for(int j = 0; j < LN; ++j) {
                            double const dev = std::abs(matY_r[c][i][j] - matY_h[iY][c][i][j]);
                            maxdev = std::max(maxdev, dev);
                            alldev += dev;
                            allval += 1.0;
                        } // j
                    } // i
                } // c
                delete[] matY_r;
            } // iY
            delete[] matY_h;
            delete[] matA_h;
            delete[] matX_h;
        } // scope

        time_chk += getTime();
        POP_RANGE(); // end of NVTX range

        std::printf("# GPU maxdev %g avgdev %g\n", maxdev, alldev/allval);
        if (maxdev > 1e-4) {
            std::printf("# Warning! GPU result has large deviations (%g) for blockDim=%d x %d\n", maxdev, LM, LN);
            correct = false; // do not show the performance of wrong results
        } else {
            std::printf("# CPU result checking with %d threads took %.3f sec\n", nthreads, time_chk);
        }
#endif // SKIP_CORRECTNESS_CHECK

        if (correct) { // print performance scope
            char const fF = FlopChar<real_t>();
            std::printf("# GPU performed %.3f T%clop in %.3f seconds\n", nFlop*1e-12, fF, time_sum);
            std::printf("# GPU performance (lm,ln,tune)=(%3d,%3d,%d) is  %.1f G%clop/sec\n",
                                            LM,LN,TUNE, nFlop*1e-9/time_sum, fF);
        } // scope

        std::printf("# %s: free GPU memory\n", __func__);
        free_gpu_memory(matX);
        free_gpu_memory(matA);
        free_gpu_memory(matY);
        free_gpu_memory(pairs_d);
        free_gpu_memory(starts_d);

        std::printf("# %s: deviceSynchronize\n", __func__);
        CCheck(hipDeviceSynchronize());
        std::printf("# %s: done\n", __func__);
        return time_avg;
    } // bench_multi

    int benchmark_blockMatrixMatrixMultiplication(int const argc, char const *const argv[]) {
        // ToDo: use control::get environment
                                 assert( 'm' == *argv[1] ); // 'multiplication' task
        char const *fnm  = (argc > 2)?           argv[2]  : "plan"; // inputfile
        char const fF    = (argc > 3)?          *argv[3]  : 'f'; // {f,F,c,C, d,D,z,Z, m,M} = float or double or mixed
        int const nrep   = (argc > 4)? std::atoi(argv[4]) : 1; // number or repetitions
        int const nsamp  = (argc > 5)? std::atoi(argv[5]) : 1; // number of sampling
        int const lm     = (argc > 6)? std::atoi(argv[6]) : 16; // block rows
        int const ln     = (argc > 7)? std::atoi(argv[7]) : lm; // block cols

        char const precision = (('d' == (fF | IgnoreCase)) || ('z' == (fF | IgnoreCase))) ? 'z'
                             : (('m' == (fF | IgnoreCase)) ? 'm' : 'c');

        // read multiplication plan from input file
        std::ifstream input(fnm, std::ifstream::in);
        if (input.fail()) {
            std::cout << argv[0] << ": error: did not find file" << std::endl; 
            exit(-3);
        } // input file not found

        std::string str;
        unsigned nnzY, nnzA, nnzX;
        input >> str >> nnzY >> nnzA >> nnzX;
        bool const info = false;
        if (info) {
            std::cout << "# nnz Y " << nnzY << std::endl;
            std::cout << "# nnz A " << nnzA << std::endl;
            std::cout << "# nnz X " << nnzX << std::endl;
        } // info

        std::vector<uint32_t> pairs;
        std::vector<uint32_t> starts;
        int64_t iY, iA, iX;
        int beta;
        int nzpr{-1};
        std::vector<int> hist(96, 0); // histogram
        int irow{-1}; //
        int64_t iYprev{-1}; // init with an impossible index value
        while (input >> iY >> iA >> iX >> beta) {
    //      std::cout << iY << " " << iA << " " << iX << " " << beta << " " << std::endl; // echo the input file structure
            if (iY != iYprev) {
                assert(0 == beta);
                starts.push_back(pairs.size()/2);
                if (-1 == iYprev) { ++hist[nzpr]; nzpr = 0; } // update histogram
                iYprev = iY;
                ++irow;
            } else {
                assert(1 == beta);
            }
    //      assert(iY == irow); // this will fail if the iY indices do not come in order
            ++nzpr; // number of non-zero entries per row
            pairs.push_back(iA); // add new small matrix-matrix-multiplications between block iA and block iX
            pairs.push_back(iX); // add new small matrix-matrix-multiplications between block iA and block iX
        } // while
        if (info) std::cout << "# found " << starts.size() << " result elements" << std::endl;
        starts.push_back(pairs.size()/2); // final
        assert(starts.size() == nnzY + 1); // we need one more for the sparse format

        if (info) {
            // show histogram about number of non-zero elements per row
            for(nzpr = 0; nzpr < 96; ++nzpr) {
                if (0 < hist[nzpr]) {
                    std::cout << "# found " << hist[nzpr] << " elements with " << nzpr << " operations" << std::endl; 
                } // nonzero
            } // nzpr
        } // info

        auto const nPairs = pairs.size()/2; // number of small matrix-matrix-multiplications
        if (info) std::cout << "# found " << nPairs << " operations" << std::endl;
#ifdef  FULLDEBUG
            std::cout << "# rows start at ";
            for(auto rs : starts) {
                std::cout << " " << rs;
            } // rs
            std::cout << std::endl;
#endif // FULLDEBUG

//      int constexpr tune = 2;
        switch (lm*1000 + ln) { // blocksize
#define call_it(REAL_t,LM,LN,DOUBLE_t,TUNE) \
            bench_multi <REAL_t,LM,LN,DOUBLE_t,TUNE> \
            (nnzY, starts.data(), nPairs, pairs.data(), nnzA, nnzX, nrep, nsamp)
#define decide_precision(LM,LN,TUNE) \
            if ('z' == precision) { call_it(double,LM,LN,double,TUNE); } else \
            if ('m' == precision) { call_it(float, LM,LN,double,TUNE); } else \
                                  { call_it(float, LM,LN,float ,TUNE); }

            // tune-parameters extracted from a comparison of TUNE={1,2,3,4,6,8} in double-performance on V100
            case   4004:  decide_precision(  4,  4, 4); break; // Lmax=1
            case   8008:  decide_precision(  8,  8, 4); break; // Lmax=1, noco
            case  16016:  decide_precision( 16, 16, 2); break; // Lmax=3
            case  32032:  decide_precision( 32, 32, 4); break; // Lmax=3, noco
            case  64064:  decide_precision( 64, 64, 2); break; // Lmax=7
            case 128128:  decide_precision(128,128, 2); break; // Lmax=7, noco

            // with a single prime factor 3
            case   6006:  decide_precision(  6,  6, 3); break;
            case  12012:  decide_precision( 12, 12, 4); break;
            case  24024:  decide_precision( 24, 24, 4); break;
            case  48048:  decide_precision( 48, 48, 6); break;
            case  96096:  decide_precision( 96, 96, 4); break;

            // rectangular cases with CUDA warp size 32
            case   4032:  decide_precision(  4, 32, 1); break;
            case   8032:  decide_precision(  8, 32, 1); break;
            case  16032:  decide_precision( 16, 32, 2); break;

#undef  decide_precision
#undef  call_it
            default : std::cout << "ERROR: Case not implemented lm = " << lm << " ln = " << std::max(lm,ln) << std::endl; return 1; 
        } // switch lm

        std::cout << "# done " << __func__ << std::endl;
        return 0; // 0:success
    } // benchmark_blockMatrixMatrixMultiplication

} // namespace GPUbench


int main(int const argc, char const *const argv[]) {

    if (argc < 2) {
        std::printf("Usage:  %s  [tfQMR/multiply]  [file]  [float/double]  "
                    "[#repetitions]  [#iterations]  [#blocksize]\n", argv[0]);
        exit(1);
    } // not enough command line args passed

    char const bench   = (argc > 1)?          *argv[1]  : 't'; // t:tfQMR, m:multiplication
    if ('m' == bench) return GPUbench::benchmark_blockMatrixMatrixMultiplication(argc, argv);

    char const *fnm    = (argc > 2)?           argv[2]  : "problem"; // inputfile
    char const flouble = (argc > 3)?        ((*argv[3]) | IgnoreCase) : 'z'; // z:double, c:float, m:mixed
    int  const nrep    = (argc > 4)? std::atoi(argv[4]) : 1; // number of repetitions
    int  const MaxIter = (argc > 5)? std::atoi(argv[5]) : 2000; // max. number of iteration

    std::printf("\n# read file '%s' as input.\n", fnm);
    bsr_t ABX[3]; // three block-sparse operators
    double tolerance{0};
    if (std::string(fnm).find("xml") != std::string::npos) {
        tolerance = tfqmrgpu_example_xml_reader::read_in(ABX, fnm);
    } else {
        tolerance = tfqmrgpu_example_reader::read_in(ABX, fnm);
    }
    std::printf("# found tolerance= %g\n", tolerance);
    std::printf("# Execute %d repetitions with max. %d iterations\n", nrep, MaxIter);
    std::printf("# requested precision= \'%c\' for LM= %d, LN= %d\n", flouble, ABX[0].fastBlockDim, ABX[1].fastBlockDim);

    return GPUbench::benchmark_tfQMRgpu_library(ABX, tolerance, MaxIter, nrep, flouble);
} // main
